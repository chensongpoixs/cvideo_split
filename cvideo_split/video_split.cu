
#include <hip/hip_runtime.h>
//#include <iostream>
//#include <opencv2/core.hpp>
//#include <opencv2/highgui.hpp>
//#include <opencv2/imgproc.hpp>
//
//__global__ void cropImage(const cv::Mat srcImg, int xOffset, int yOffset, int width, int height) {
//     ��ȡ��ǰ�߳�����
//    const int threadIdx = blockDim.x * blockIdx.y + blockIdx.x;
//
//    if (threadIdx >= width * height) return;
//
//     ����ԭʼͼ����ÿ�����ص��λ��
//    int row = threadIdx / width;
//    int col = threadIdx % width;
//
//     ����ƫ����������ͼ���ϵ�λ��
//    int newRow = row - yOffset;
//    int newCol = col - xOffset;
//
//     �ж���λ���Ƿ�Խ��
//    if ((newRow >= 0 && newRow < height) && (newCol >= 0 && newCol < width)) {
//         ��ԭʼͼ���ϵ�����ֵ���Ƶ���ͼ������Ӧλ��
//        dstImg.at<uchar>(row, col) = srcImg.at<uchar>(newRow, newCol);
//    }
//    else {
//         ���ó����߽粿��Ϊ��ɫ
//        dstImg.at<uchar>(row, col) = 0;
//    }
//}
//
//int main() {
//     ����Դͼ��
//    cv::Mat srcImg = cv::imread("input_image.jpg", cv::IMREAD_GRAYSCALE);
//
//     ����ü��������ʼ����ʹ�С
//    int xOffset = 100;
//    int yOffset = 50;
//    int width = 300;
//    int height = 200;
//
//     ����Ŀ��ͼ��
//    cv::Mat dstImg(srcImg.rows, srcImg.cols, CV_8UC1);
//
//     ����CUDA�ڴ�
//    uchar* devSrcPtr;
//    uchar* devDstPtr;
//    size_t imgSize = srcImg.total() * sizeof(uchar);
//    cudaMalloc((void**)&devSrcPtr, imgSize);
//    cudaMalloc((void**)&devDstPtr, imgSize);
//
//     ��Դͼ��������ڴ洫�䵽�豸�ڴ�
//    cudaMemcpy(devSrcPtr, srcImg.data, imgSize, cudaMemcpyHostToDevice);
//
//     ����������߳̿�Ĵ�С
//    cv::dim3 gridSize((width + BLOCK_SIZE - 1) / BLOCK_SIZE, (height + BLOCK_SIZE - 1) / BLOCK_SIZE);
//    cv::dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
//
//     ����CUDA kernel��������ͼ��ü�
//    cropImage << <gridSize, blockSize >> > (devSrcPtr, xOffset, yOffset, width, height);
//
//     �ȴ������������
//    cudaDeviceSynchronize();
//
//     ��������豸�ڴ洫��������ڴ�
//    cudaMemcpy(dstImg.data, devDstPtr, imgSize, cudaMemcpyDeviceToHost);
//
//     ����ü����ͼ��
//    cv::imwrite("output_image.jpg", dstImg);
//
//     ���CUDA�ڴ�
//    cudaFree(devSrcPtr);
//    cudaFree(devDstPtr);
//
//    return 0;
//}