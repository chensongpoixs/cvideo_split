
#include <hip/hip_runtime.h>
﻿//#include <iostream>
//#include <opencv2/core.hpp>
//#include <opencv2/highgui.hpp>
//#include <opencv2/imgproc.hpp>
//
//__global__ void cropImage(const cv::Mat srcImg, int xOffset, int yOffset, int width, int height) {
//     获取当前线程索引
//    const int threadIdx = blockDim.x * blockIdx.y + blockIdx.x;
//
//    if (threadIdx >= width * height) return;
//
//     计算原始图像上每个像素点的位置
//    int row = threadIdx / width;
//    int col = threadIdx % width;
//
//     根据偏移量调整新图像上的位置
//    int newRow = row - yOffset;
//    int newCol = col - xOffset;
//
//     判断新位置是否越界
//    if ((newRow >= 0 && newRow < height) && (newCol >= 0 && newCol < width)) {
//         将原始图像上的像素值复制到新图像上相应位置
//        dstImg.at<uchar>(row, col) = srcImg.at<uchar>(newRow, newCol);
//    }
//    else {
//         设置超出边界部分为黑色
//        dstImg.at<uchar>(row, col) = 0;
//    }
//}
//
//int main() {
//     加载源图像
//    cv::Mat srcImg = cv::imread("input_image.jpg", cv::IMREAD_GRAYSCALE);
//
//     定义裁剪区域的起始坐标和大小
//    int xOffset = 100;
//    int yOffset = 50;
//    int width = 300;
//    int height = 200;
//
//     创建目标图像
//    cv::Mat dstImg(srcImg.rows, srcImg.cols, CV_8UC1);
//
//     配置CUDA内存
//    uchar* devSrcPtr;
//    uchar* devDstPtr;
//    size_t imgSize = srcImg.total() * sizeof(uchar);
//    cudaMalloc((void**)&devSrcPtr, imgSize);
//    cudaMalloc((void**)&devDstPtr, imgSize);
//
//     将源图像从主机内存传输到设备内存
//    cudaMemcpy(devSrcPtr, srcImg.data, imgSize, cudaMemcpyHostToDevice);
//
//     定义网格和线程块的大小
//    cv::dim3 gridSize((width + BLOCK_SIZE - 1) / BLOCK_SIZE, (height + BLOCK_SIZE - 1) / BLOCK_SIZE);
//    cv::dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
//
//     运行CUDA kernel函数进行图像裁剪
//    cropImage << <gridSize, blockSize >> > (devSrcPtr, xOffset, yOffset, width, height);
//
//     等待所有任务完成
//    cudaDeviceSynchronize();
//
//     将结果从设备内存传输回主机内存
//    cudaMemcpy(dstImg.data, devDstPtr, imgSize, cudaMemcpyDeviceToHost);
//
//     保存裁剪后的图像
//    cv::imwrite("output_image.jpg", dstImg);
//
//     清除CUDA内存
//    cudaFree(devSrcPtr);
//    cudaFree(devDstPtr);
//
//    return 0;
//}